#include "hip/hip_runtime.h"
// SPM Max Pooling CUDA
// Author: Vic Chan
// Date: 2018/5/21
#ifdef __cplusplus
extern "C" {
#endif

#include <stdio.h>
#include "spmmax_pooling_kernel.h"

__global__ void spmmax_pooling_forward(int batch_size, int num_grids, int feature_size,
                                       int num_rois, float* x_data, float* shapes_data, float* rois_data,
                                       float* output_data, int* max_ids_data, float* spm){

  int thread_idx = threadIdx.x + blockIdx.x*blockDim.x;
  if (thread_idx < num_rois * num_grids * feature_size) {
    int roi_id = thread_idx/(num_grids * feature_size);
    int grid_id = (thread_idx - roi_id * num_grids * feature_size)/feature_size;
    int feature_id = thread_idx - roi_id * num_grids * feature_size - grid_id * feature_size;

    int batch_id = (int)rois_data[roi_id*5];
    float center_x = (rois_data[roi_id*5+1] + rois_data[roi_id*5+3])/(2*shapes_data[batch_id*2+0]);
    float center_y = (rois_data[roi_id*5+2] + rois_data[roi_id*5+4])/(2*shapes_data[batch_id*2+1]);

    if (center_x >= spm[grid_id*4+0] && center_x < spm[grid_id*4+1]
        && center_y >= spm[grid_id*4+2] && center_y < spm[grid_id*4+3]) {
      int idx = batch_id*num_grids*feature_size + grid_id * feature_size + feature_id;
      if (x_data[roi_id*feature_size + feature_id] > output_data[idx]) {
        atomicExch(output_data+idx, x_data[roi_id*feature_size + feature_id]);
        atomicExch(max_ids_data+idx, roi_id);
      }
    }
  }
  __syncthreads();
}

__global__ void spmmax_pooling_backward(int batch_size, int num_grids, int feature_size, int num_rois,
                                        float* grad_input_data,float* grad_output_data, int* max_ids_data) {

  int thread_idx = threadIdx.x + blockIdx.x*blockDim.x;
  if (thread_idx < batch_size * num_grids * feature_size) {
    int batch_id = thread_idx / (num_grids * feature_size);
    int grid_id = (thread_idx - (num_grids * feature_size * batch_id)) / feature_size;
    int feature_id = thread_idx - num_grids * feature_size * batch_id - feature_size * grid_id;

    int idx = batch_id*num_grids*feature_size + grid_id * feature_size + feature_id;
    if (max_ids_data[idx] == -1) {
      atomicAdd(grad_output_data+max_ids_data[idx]*feature_size + feature_id, grad_input_data[idx]);
  }


  }
  __syncthreads();
}

int spmmax_pooling_forward_kernel(int batch_size, int num_grids, int feature_size, int num_rois, float* x_data,
                                  float* shapes_data, float* rois_data, float* output_data, int* max_ids_data) {
  int output_size = num_rois * num_grids * feature_size;
  hipError_t err;

  float spm[32] = {0, 1, 0, 1, 0, 0.5, 0, 0.5, 0, 0.5, 0.5, 1, 0.5, 1, 0, 0.5, 0.5,
                              1, 0.5, 1, 0, 1, 0, 0.33, 0, 1, 0.33, 0.67, 0, 1, 0.67, 1};

  const int kThreadsPerBlock = 1024;
  dim3 threads(kThreadsPerBlock);
  int block = (output_size + kThreadsPerBlock - 1)/kThreadsPerBlock;
  if (block == 0)
    block = 1;
  dim3 blocks(block);
  spmmax_pooling_forward<<<blocks, threads>>>(batch_size, num_grids, feature_size, num_rois, x_data,
      shapes_data,rois_data, output_data, max_ids_data, spm);
  err = hipGetLastError();
  if(hipSuccess != err)
  {
    fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
    exit( -1 );
  }

  return 1;
}

int spmmax_pooling_backward_kernel(int batch_size, int num_grids, int feature_size, int num_rois, float* grad_input_data,
                                   float* grad_output_data, int* max_ids_data) {

  const int kThreadsPerBlock = 1024;
  int output_size = batch_size * num_grids * feature_size;
  hipError_t err;
  dim3 threads(kThreadsPerBlock);
  int block = (output_size + kThreadsPerBlock - 1)/kThreadsPerBlock;
  if (block == 0)
    block = 1;
  dim3 blocks(block);
  spmmax_pooling_backward<<<blocks, threads>>>(batch_size, num_grids, feature_size, num_rois, grad_input_data,
      grad_output_data, max_ids_data);

  err = hipGetLastError();
  if(hipSuccess != err)
  {
    fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
    exit(-1);
  }

  return 1;
}


#ifdef __cplusplus
}
#endif
